#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <ctime>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#include "dxt1.h"
#include "types.h"
#include "colors.h"

hipError_t compressWithCuda(BYTE*, BYTE*, int, int);

__global__ void compressDXT1(BYTE *dev_image, BYTE *dev_compressedImage, long imageSize, int maxDXT1BlocksPerRow, int width)
{
	long currBlockIdx = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	long blockRow = currBlockIdx / maxDXT1BlocksPerRow;
	long blockCol = currBlockIdx % maxDXT1BlocksPerRow;

	long outBuffOffset = currBlockIdx*8;
	long imageOffset = blockRow* width* 16 + blockCol *16;

	if (imageOffset  > imageSize - 64) {
		return;
	}

	BYTE* blockStart = dev_image + imageOffset;
	BYTE* outBuff = dev_compressedImage + outBuffOffset;
	BYTE block[64];
	BYTE minColor[3];
	BYTE maxColor[3];
	DWORD colorIndices;

	extractBlock(blockStart, width, block);
	getMinMaxColors(block, minColor, maxColor);
	getColorIndices(block, minColor, maxColor, &colorIndices);

	writeWord(outBuff, dev_rgb888_to_rgb565(maxColor));
	writeWord(outBuff, dev_rgb888_to_rgb565(minColor));
	writeDoubleWord(outBuff, colorIndices);
}

int main()
{
	char inPath[] = "./images/pretty.png";
	char outPath[] = "./output/out.png";
	int width, height, bytesPerPixel;
	long pixelCount;
	BYTE *imageData;
	hipError_t cudaStatus;

	BYTE *oldImageData = stbi_load(inPath, &width, &height, &bytesPerPixel, 0);
	if (oldImageData == NULL) {
		printf("Invalid image!");
		return  1;
	}
	if (width % 4 != 0 || height % 4 != 0) {
		printf("Invalid dimensions");
		return  1;
	}
	if (width > 32768 || height > 32768) {
		printf("Max image dimensions: 32768x32768");
		return  1;
	}

	pixelCount = width*height;

	// Check if image is RGB and convert to RGBA
	imageData = (BYTE *)malloc(pixelCount * 4);
	if (bytesPerPixel == 3) {
		bytesPerPixel = 4;
		rgb_to_rgba_image(oldImageData, imageData, pixelCount);
	} else if (bytesPerPixel == 4) {
		memcpy(imageData, oldImageData, pixelCount*bytesPerPixel);
	} else {
		printf("Invalid pixel size! %d %d %d", bytesPerPixel, width, height);
		return 1;
	}

	BYTE *compressedImage = (BYTE*)malloc(pixelCount * 4 / 8);
	BYTE* decompressedImage = (BYTE*)malloc(pixelCount * 4);

	clock_t compressStart = clock();
	// Compress image DXT1(8:1 compression ratio)
	cudaStatus = compressWithCuda(imageData, compressedImage, width, height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Compress failed!");
		return 1;
	}
	clock_t compressEnd = clock();


	// Decompress image DXT1
	DecompressImageDXT1(width, height, compressedImage, decompressedImage);

	clock_t decompressEnd = clock();

	printf("compress: %.3fsec\n", double(compressEnd - compressStart) / CLOCKS_PER_SEC);
	printf("decompress: %.3fsec\n", double(decompressEnd - compressEnd) / CLOCKS_PER_SEC);
	printf("total: %.3fsec\n", double(decompressEnd - compressStart) / CLOCKS_PER_SEC);

	//if (stbi_write_jpg(outPath, width, height, bytesPerPixel, imageData, 100)) {
	if (stbi_write_png(outPath, width, height, bytesPerPixel, decompressedImage, width*bytesPerPixel)) {
		printf("WRITE SUCCESS!");
	} else {
		printf("WRITE ERROR!");
	}

	stbi_image_free(imageData);
	stbi_image_free(oldImageData);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t compressWithCuda(BYTE *image, BYTE *compressedImage, int width, int height)
{
	time_t totalStart = clock();

	BYTE *dev_image = 0;
	BYTE *dev_compressedImage = 0;
	long imageSize = width*height*4;
	int maxDXT1BlocksPerRow = width / 4;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_image, imageSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_compressedImage, imageSize / 8);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_image, image, imageSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	
	int threadsCount = 256;
	dim3 blocks(1,1,1);

	long totalBlocks = ceil(height*width / 16 / float(threadsCount));
	if (totalBlocks <= 1024) {
		blocks.x = totalBlocks;
	} else {
		blocks.x = 1024;
		blocks.y = ceil(totalBlocks / 1024.f);
	}

	time_t compressStart = clock();

    // Launch a kernel on the GPU with one thread for each element.
    compressDXT1<<<blocks, threadsCount>>>(dev_image, dev_compressedImage, imageSize, maxDXT1BlocksPerRow, width);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	time_t compressEnd = clock();

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(compressedImage, dev_compressedImage, width*height*4 / 8, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	time_t totalEnd = clock();

	printf("Compress time: %.3lfs\n", double(compressEnd - compressStart) / CLOCKS_PER_SEC);
	printf("Total compress time(+memory allocation): %.3lfs\n", double(totalEnd - totalStart) / CLOCKS_PER_SEC);

Error:
    hipFree(dev_compressedImage);
    hipFree(dev_image);
    
    return cudaStatus;
}